#include <hip/hip_runtime.h>
#include <sys/mman.h>
#include <stdlib.h>
#include <sys/time.h>
#include <stdint.h>
#include <iostream>
#include <emmintrin.h>
#include <cassert>
#include <climits>
#include <string.h>
#include <algorithm>
#include <vector>
#include <random>
#include <utility>
#include <regex>

#define QUERIES_PER_TRIAL (50 * 1000 * 1000)
#define NUM_OF_BLOCKS 1024

const unsigned K=3;

struct LeafEntry {
    int32_t key;
    uint64_t value;
};

void* malloc_huge_cuda(size_t size) {
    void* p;
    hipMalloc(&p, size);
    return p;
}


inline unsigned pow16(unsigned exponent) {
    // 16^exponent
    return 1<<(exponent<<2);
}

inline unsigned median(unsigned i,unsigned j) {
    return i+(j-1-i)/2;
}


__global__ void searchKernel(const int32_t* v, const int32_t* keys,  int *result, unsigned scale, unsigned index) {
    unsigned key_i = blockIdx.x+NUM_OF_BLOCKS*index;
    if (key_i>=QUERIES_PER_TRIAL)
        return;
    int key_q = keys[key_i];
    const unsigned commonAncesterArray[] = {16,16,16,16,16,16,16,3,1,4,0,5,2,6,16};
    unsigned simd_lane = threadIdx.x;
    unsigned const ancestor = commonAncesterArray[simd_lane];
    __shared__ int child_index;
    __shared__ int shared_gt[16];
    unsigned levelOffset = 0;
    for (int i=0;i<4;++i) {
        size_t addr = ((1 << (4 * i)) - 1) + levelOffset * 16;

//        printf("Thread %d is running\n", idx);
        int32_t v_node = *(v+addr+simd_lane);
//        printf("Thread %d is running after v\n", idx);
        int32_t gt = (key_q>v_node);
        shared_gt[simd_lane] = gt;
//        printf("Thread %d is running after v\n", idx);
        __syncthreads();

        int32_t next_gt = shared_gt[simd_lane+1];
        if (threadIdx.x == 7) {
            if(!gt)
                child_index = 0;
        }
        if (threadIdx.x >= 7 && threadIdx.x<14) {
            if(gt && next_gt==0) {
                child_index = shared_gt[commonAncesterArray[threadIdx.x]]+simd_lane*2-13;
            }
        }
        __syncthreads();
        levelOffset = levelOffset * 16 + child_index;
    }


    unsigned offset = 69904 + levelOffset*scale;
    unsigned pos = levelOffset;
    levelOffset = 0;
    unsigned pageOffset = 0;

    for (int j=0;j<3;++j) {
        size_t addr = offset + (2^(4*j)-1) + levelOffset * 16;
        int32_t v_node = v[addr+simd_lane];

        int32_t gt = (key_q>v_node);
        shared_gt[simd_lane] = gt;
        __syncthreads();

        int32_t next_gt = shared_gt[simd_lane+1];
        if (threadIdx.x == 7 && !gt) {
            child_index = 0;
        }
        if (threadIdx.x >= 7) {
            if(gt && !next_gt) {
                child_index = shared_gt[commonAncesterArray[threadIdx.x]]+simd_lane*2-13;
                if (j==2) {
                    levelOffset = levelOffset * 16 + child_index;
                    int res = ((pos << (4*3)) | levelOffset);
                    result[key_i] = res;
                }
            }
        }
        __syncthreads();
        levelOffset = levelOffset * 16 + child_index;
    }

}

void cudaSearch(std::vector<int>& queries, const int32_t* fast, unsigned scale, int* res) {
    int* deviceData;
    int sizeInByte = queries.size()*sizeof(int);
    hipMalloc(&deviceData, sizeInByte);
    hipMemcpy(deviceData, queries.data(), sizeInByte, hipMemcpyHostToDevice);
    const int numStreams = 648;
    int * check;
    hipMalloc(&check, sizeInByte);

    hipStream_t streams[numStreams];
    for(int i=0;i<numStreams;++i) {
        hipStreamCreate(&streams[i]);
    }

    for(size_t i=0;i<queries.size()/NUM_OF_BLOCKS;++i) {
        int streamIndex = i % numStreams;
        int key = queries[i];
        searchKernel<<<NUM_OF_BLOCKS, 15, 0, streams[streamIndex]>>>(fast, deviceData, check, scale, i);
    }

    for (int i = 0; i < numStreams; ++i) {
        hipStreamSynchronize(streams[i]);
//        cudaStreamDestroy(streams[i]);
    }

//    uint64_t hostCheck = 0;
    hipMemcpy(res, check, sizeInByte, hipMemcpyDeviceToHost);

//    for (int i=0;i<queries.size();++i) {
//        hostCheck += check[i];
//    }

    for (int i = 0; i < numStreams; ++i) {
//        cudaStreamSynchronize(streams[i]);
        hipStreamDestroy(streams[i]);
    }

    hipFree(check);
    hipFree(deviceData);
}



int32_t * pinCuda(int32_t *fast, unsigned n) {
    void* add = (malloc_huge_cuda(n*sizeof(int32_t)));
    hipMemcpy(add, fast, sizeof(int) * n, hipMemcpyHostToDevice);
    return (int32_t *)add;
}


